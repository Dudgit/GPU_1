#include "hip/hip_runtime.h"
// Include Cuda libraries, 'cause I use Visual Studio
#include "hip/hip_runtime.h"
#include ""
//Refference functions
#include "vector_mx_mul.h"
#include "cpp_functions.h"

//Constant values


__global__ void mx_vec_gpu(float* result_mx, float* input_vector, float* input_mx, int N)
{

    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ float Atmp[MBS * MBS];
    __shared__ float Btmp[MBS*MBS];


    float sum = 0.f;

    for (int K = 0; K < N / MBS; ++K)
    {
        Atmp[threadIdx.y * MBS + threadIdx.x] = input_mx[y * N + (K * MBS + threadIdx.x)];
        Btmp[threadIdx.y * MBS+threadIdx.x] = input_vector[(K * MBS + threadIdx.y)];
        
        __syncthreads();
        for (int k = 0; k < MBS; ++k)
        {
            sum += Atmp[threadIdx.y * MBS + k] * Btmp[k * MBS + threadIdx.x];
        }
        __syncthreads();
    }
    result_mx[y] = sum;
}

float do_Cuda(std::vector<float>& A, std::vector<float>& B, std::vector<float>& C0, std::vector<float>& C1, std::vector<float>& C2)
{

    float* pA = nullptr;
    float* pB = nullptr;
    float* pC2 = nullptr;

    hipEvent_t evt[2];
    for (auto& e : evt) { hipEventCreate(&e); }

    hipError_t err = hipSuccess;

    err = hipMalloc((void**)&pA, N * sizeof(float));
    if (err != hipSuccess) { std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipMalloc((void**)&pB, N * N * sizeof(float));
    if (err != hipSuccess) { std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipMalloc((void**)&pC2, N * sizeof(float));
    if (err != hipSuccess) { std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipMemcpy(pA, A.data(), N * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) { std::cout << "Error copying memory to device: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipMemcpy(pB, B.data(), N * N * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) { std::cout << "Error copying memory to device: " << hipGetErrorString(err) << "\n"; return -1; }

    {
        dim3 dimGrid(n_blocks, n_blocks);
        dim3 dimBlock(block_sz, block_sz);
        hipEventRecord(evt[0]);
        mx_vec_gpu << <dimGrid, dimBlock >> > (pC2, pA, pB, N);
        err = hipGetLastError();
        if (err != hipSuccess) { std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << "\n"; return -1; }
        hipEventRecord(evt[1]);
    }
    err = hipMemcpy(C2.data(), pC2, N * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) { std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipFree(pA);
    if (err != hipSuccess) { std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipFree(pB);
    if (err != hipSuccess) { std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipFree(pC2);
    if (err != hipSuccess) { std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

    hipEventSynchronize(evt[1]);
    float dt = 0.0f;//milliseconds
    hipEventElapsedTime(&dt, evt[0], evt[1]);
    for (auto& e : evt) { hipEventDestroy(e); }
    return dt;
}

int main()
{
    std::vector<float> A(N);
    std::vector<float> B(N * N);
    std::vector<float> C0(N);
    std::vector<float> C1(N);
    std::vector<float> C2(N);
    fill_values(A, B, C0, C1, C2);
    float cuda_run_time = do_Cuda(A, B, C0, C1, C2);

    auto t0 = std::chrono::high_resolution_clock::now();
    vector_mx_naive(C0, A, B, N);
    auto t1 = std::chrono::high_resolution_clock::now();
    vector_mx_alg(C1, A, B, N);
    auto t2 = std::chrono::high_resolution_clock::now();
    checker(C0, C1, "C0 vs C1","CPU Naive","CPU improved");
    checker(C0, C2, "C0 vs C2", "CPU Naive", "GPU Naive");
  

    std::cout << "CPU naive    Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count() / 1000.0f << " ms\n";
    std::cout << "CPU improved Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() / 1000.0f << " ms\n";
    std::cout << "GPU improved Computation took: " << cuda_run_time << " ms.\n";
 
    return 0;
}
